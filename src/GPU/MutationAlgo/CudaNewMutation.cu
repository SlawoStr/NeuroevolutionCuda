#include "hip/hip_runtime.h"
#include "CudaNewMutation.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void newMutationKernel(float* weight, float mutationProbability, float geneMutationProbability, int weightPerModel, int taskNumber, hiprandState* state)
{
	__shared__ float mutationProba;
	hiprandState threadState = state[threadIdx.x + blockIdx.x * blockDim.x];
	for (int i = blockIdx.x; i < taskNumber; i += gridDim.x)
	{
		if (threadIdx.x == 0)
		{
			mutationProba = randomFloat(threadState, 0.0f, 1.0f);
		}
		__syncthreads();
		if (mutationProba < mutationProbability)
		{
			// Loop over genes
			for (int j = threadIdx.x; j < weightPerModel; j += blockDim.x)
			{
				if (randomFloat(threadState, 0.0f, 1.0f) < geneMutationProbability)
				{
					weight[i * weightPerModel + j] = randomFloat(threadState, -1.0f, 1.0f);
				}
			}
		}
		__syncthreads();
	}
	state[threadIdx.x + blockIdx.x * blockDim.x] = threadState;
}


void CudaNewMutation::runMutation(thrust::device_vector<float>& newWeights, int weightPerModel, int parentModel, hiprandState* state, unsigned blockNumber, unsigned threadNumber)
{


}
