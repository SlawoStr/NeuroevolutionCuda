#include "hip/hip_runtime.h"
#include "CudaNewMutation.cuh"
#include "hip/hip_runtime.h"
#include ""

__forceinline__ __device__ float randomFloat2(hiprandState& state, float minValue, float maxValue)
{
	return hiprand_uniform(&state) * (maxValue - minValue) + minValue;
}

__global__ void newMutationKernel(float* weight, float mutationProbability, float geneMutationProbability, int weightPerModel, int taskNumber, hiprandState* state)
{
	__shared__ float mutationProba;
	hiprandState threadState = state[threadIdx.x + blockIdx.x * blockDim.x];
	for (int i = blockIdx.x; i < taskNumber; i += gridDim.x)
	{
		if (threadIdx.x == 0)
		{
			mutationProba = randomFloat(threadState, 0.0f, 1.0f);
		}
		__syncthreads();
		if (mutationProba < mutationProbability)
		{
			int weightOffset = i * weightPerModel;
			// Loop over genes
			for (int j = threadIdx.x; j < weightPerModel; j += blockDim.x)
			{
				if (randomFloat(threadState, 0.0f, 1.0f) < geneMutationProbability)
				{
					weight[weightOffset + j] = randomFloat(threadState, -1.0f, 1.0f);
				}
			}
		}
		__syncthreads();
	}
	state[threadIdx.x + blockIdx.x * blockDim.x] = threadState;
}


void CudaNewMutation::runMutation(thrust::device_vector<float>& newWeights, int weightPerModel, int parentNumber, hiprandState* state, unsigned blockNumber, unsigned threadNumber)
{
	newMutationKernel << <std::min(blockNumber, static_cast<unsigned>(parentNumber)), threadNumber >> >
		(thrust::raw_pointer_cast(newWeights.data()), m_mutationProbability, m_geneMutationProbability, weightPerModel, parentNumber, state);
}
