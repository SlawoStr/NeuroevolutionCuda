#include "hip/hip_runtime.h"
#include "CudaProcMutation.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void procMutationKernel(float* weight, float mutationProbability, float geneMutationProbability, int weightPerModel, int taskNumber, hiprandState* state, float minVal, float maxVal)
{
	__shared__ float mutationProba;
	hiprandState threadState = state[threadIdx.x + blockIdx.x * blockDim.x];
	for (int i = blockIdx.x; i < taskNumber; i += gridDim.x)
	{
		if (threadIdx.x == 0)
		{
			mutationProba = randomFloat(threadState, 0.0f, 1.0f);
		}
		__syncthreads();
		if (mutationProba < mutationProbability)
		{
			// Loop over genes
			for (int j = threadIdx.x; j < weightPerModel; j += blockDim.x)
			{
				if (randomFloat(threadState, 0.0f, 1.0f) < geneMutationProbability)
				{
					weight[i * weightPerModel + j] *= randomFloat(threadState, minVal, maxVal);
				}
			}
		}
		__syncthreads();
	}
	state[threadIdx.x + blockIdx.x * blockDim.x] = threadState;
}



void CudaProcMutation::runMutation(thrust::device_vector<float>& newWeights, int weightPerModel, int parentModel, hiprandState* state, unsigned blockNumber, unsigned threadNumber)
{

}