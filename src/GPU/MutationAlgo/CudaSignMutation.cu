#include "hip/hip_runtime.h"
#include "CudaSignMutation.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void signMutationKernel(float* weight, float mutationProbability, float geneMutationProbability, int weightPerModel, int taskNumber, hiprandState* state)
{
	__shared__ float mutationProba;
	hiprandState threadState = state[threadIdx.x + blockIdx.x * blockDim.x];
	for (int i = blockIdx.x; i < taskNumber; i += gridDim.x)
	{
		if (threadIdx.x == 0)
		{
			mutationProba = randomFloat(threadState, 0.0f, 1.0f);
		}
		__syncthreads();
		if (mutationProba < mutationProbability)
		{
			// Loop over genes
			for (int j = threadIdx.x; j < weightPerModel; j += blockDim.x)
			{
				if (randomFloat(threadState, 0.0f, 1.0f) < geneMutationProbability)
				{
					weight[i * weightPerModel + j] *= -1;
				}
			}
		}
		__syncthreads();
	}
	state[threadIdx.x + blockIdx.x * blockDim.x] = threadState;
}

void CudaSignMutation::runMutation(thrust::device_vector<float>& newWeights, int weightPerModel, int parentNumber, hiprandState* state, unsigned blockNumber, unsigned threadNumber)
{
	signMutationKernel << <std::min(blockNumber, static_cast<unsigned>(parentNumber)), threadNumber >> >
		(thrust::raw_pointer_cast(newWeights.data()), m_mutationProbability, m_geneMutationProbability, weightPerModel, parentNumber, state);
}
