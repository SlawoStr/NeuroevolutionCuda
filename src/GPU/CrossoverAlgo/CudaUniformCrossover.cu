#include "hip/hip_runtime.h"
#include "CudaUniformCrossover.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void crossoverUniformKernel(float* weight, float crossoverProbability, int weightPerModel, int taskNumber, hiprandState* state)
{
	__shared__ float crossProba;

	for (int i = blockIdx.x; i < taskNumber; i += gridDim.x)
	{
		if (threadIdx.x == 0)
		{
			crossProba = randomFloat(state[threadIdx.x + blockIdx.x * blockDim.x], 0.0f, 1.0f);
		}
		__syncthreads();
		if (crossProba < crossoverProbability)
		{
			int lhsOffset = i * 2 * weightPerModel;
			int rhsOffset = (i * 2 + 1) * weightPerModel;
			for (int j = threadIdx.x; j < weightPerModel; j += blockDim.x)
			{
				if (randomFloat(state[threadIdx.x + blockIdx.x * blockDim.x], 0.0f, 1.0f) < 0.5f)
				{
					float value = weight[lhsOffset + j];
					weight[lhsOffset + j] = weight[rhsOffset + j];
					weight[rhsOffset + j] = value;
				}
			}
		}
		__syncthreads();
	}
}

void CudaUniformCrossover::runCrossover(thrust::device_vector<float>& newWeights, int weightPerModel, int parentPairNumber, hiprandState* state, unsigned blockNumber, unsigned threadNumber)
{
	int optBlockNumber = std::min(static_cast<unsigned>(parentPairNumber), blockNumber);
	crossoverUniformKernel << < optBlockNumber, threadNumber >> > (thrust::raw_pointer_cast(newWeights.data()), m_crossProbability, weightPerModel, parentPairNumber, state);
}