#include "hip/hip_runtime.h"
#include "CudaSPCCrossover.cuh"
#include "hip/hip_runtime.h"
#include ""


__global__ void crossoverSPCKernel(float* weight, float crossoverProbability, int weightPerModel, int taskNumber, hiprandState* state)
{
	__shared__ int crossPoint;
	__shared__ float crossProba;

	for (int i = blockIdx.x; i < taskNumber; i += gridDim.x)
	{
		if (threadIdx.x == 0)
		{
			crossProba = randomFloat(state[blockIdx.x], 0.0f, 1.0f);
			crossPoint = randomFloat(state[blockIdx.x], 1, weightPerModel - 1);
		}
		__syncthreads();

		if (crossProba < crossoverProbability)
		{
			int lhsOffset = i * 2 * weightPerModel;
			int rhsOffset = (i * 2 + 1) * weightPerModel;
			for (int j = threadIdx.x; j < crossPoint; j += blockDim.x)
			{
				float value = weight[lhsOffset + j];
				weight[lhsOffset + j] = weight[rhsOffset + j];
				weight[rhsOffset + j] = value;
			}
		}
		__syncthreads();
	}
}

void CudaSPCCrossover::runCrossover(thrust::device_vector<float>& newWeights, int weightPerModel, int parentPairNumber, hiprandState* state, unsigned blockNumber, unsigned threadNumber)
{
	int optBlockNumber = std::min(static_cast<unsigned>(parentPairNumber), threadNumber * blockNumber);
	crossoverSPCKernel << < optBlockNumber, threadNumber >> > (thrust::raw_pointer_cast(newWeights.data()), m_crossProbability, weightPerModel, parentPairNumber, state);
}