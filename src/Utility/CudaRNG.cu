#include "hip/hip_runtime.h"
#include "CudaRNG.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "CudaError.h"

////////////////////////////////////////////////////////////
__global__ void kernelInitRandomState(unsigned seed, hiprandState* state)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, tid, 0, &state[tid]);
}

////////////////////////////////////////////////////////////
__device__ int randomInt(hiprandState& state, int minValue, int maxValue)
{
	maxValue -= 1;
	float randomValue = hiprand_uniform(&state);
	randomValue *= (maxValue - minValue + 0.999999);
	randomValue += minValue;
	return (int)truncf(randomValue);
}

////////////////////////////////////////////////////////////
__device__ float randomFloat(hiprandState& state, float minValue, float maxValue)
{
	float randomValue = hiprand_uniform(&state);
	return randomValue * (maxValue - minValue) + minValue;
}

////////////////////////////////////////////////////////////
__global__ void kernelRandomInt(int minValue, int maxValue, size_t size, int* d_arr, hiprandState* state)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;
	// Copy state to local memory for efficiency
	hiprandState localState = state[tid];
	for (int i = tid; i < size; i += stride)
	{
		d_arr[i] = randomInt(localState, minValue, maxValue);
	}
	// Copy state back to global memory
	state[tid] = localState;
}

////////////////////////////////////////////////////////////
__global__ void kernelRandomFloat(float minValue, float maxValue, size_t size, float* d_arr, hiprandState* state)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;
	// Copy state to local memory for efficiency
	hiprandState localState = state[tid];
	// Generate pseudo-random floats
	for (int i = tid; i < size; i += stride)
	{
		d_arr[i] = randomFloat(localState, minValue, maxValue);
	}
	// Copy state back to global memory
	state[tid] = localState;
}

////////////////////////////////////////////////////////////
hiprandState* generateRandomStates(unsigned blockNumber, unsigned threadNumber, unsigned seed)
{
	hiprandState* state{ nullptr };
	checkCudaErrors(hipMalloc(&state, blockNumber * threadNumber * sizeof(hiprandState)));
	kernelInitRandomState << <blockNumber, threadNumber >> > (seed, state);
	checkCudaErrors(hipPeekAtLastError());
	return state;
}

////////////////////////////////////////////////////////////
void randomFloat(unsigned blockNumber, unsigned threadNumber, float min, float max, size_t size, float* d_arr, hiprandState* state)
{
	kernelRandomFloat << < blockNumber, threadNumber >> > (min, max, size, d_arr, state);
	checkCudaErrors(hipPeekAtLastError());
}

////////////////////////////////////////////////////////////
float* randomFloat(unsigned blockNumber, unsigned threadNumber, float min, float max, size_t size, hiprandState* state)
{
	float* d_arr{ nullptr };
	checkCudaErrors(hipMalloc(&d_arr, sizeof(float) * size));
	kernelRandomFloat << < blockNumber, threadNumber >> > (min, max, size, d_arr, state);
	checkCudaErrors(hipPeekAtLastError());
	return d_arr;
}

////////////////////////////////////////////////////////////
void randomInt(unsigned blockNumber, unsigned threadNumber, int min, int max, size_t size, int* d_arr, hiprandState* state)
{
	kernelRandomInt << < blockNumber, threadNumber >> > (min, max, size, d_arr, state);
	checkCudaErrors(hipPeekAtLastError());
}

////////////////////////////////////////////////////////////
int* randomInt(unsigned blockNumber, unsigned threadNumber, int min, int max, size_t size, hiprandState* state)
{
	int* d_arr{ nullptr };
	checkCudaErrors(hipMalloc(&d_arr, sizeof(int) * size));
	kernelRandomInt << < blockNumber, threadNumber >> > (min, max, size, d_arr, state);
	checkCudaErrors(hipPeekAtLastError());
	return d_arr;
}
